#include <cutlass/cutlass.h>
#include <cutlass/half.h>
#include <cutlass/layout/matrix.h>
#include <cutlass/arch/arch.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/threadblock/default_mma_core_sm80.h>
#include <cutlass/epilogue/thread/linear_combination.h>

#include <hip/hip_runtime.h>
#include <stdexcept>
#include "lucidia_gemm/common.h"

namespace lucidia_gemm {

// FP16 inputs, FP32 accumulate, FP32 output
using ElementInputA = cutlass::half_t;
using ElementInputB = cutlass::half_t;
using ElementAccumulator = float;
using ElementOutput = float;

using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::RowMajor;
using LayoutC = cutlass::layout::RowMajor;

using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput, /*ElementsPerAccess*/ 4, ElementAccumulator, ElementAccumulator>;

using Gemm = cutlass::gemm::device::Gemm<
    ElementInputA, LayoutA,
    ElementInputB, LayoutB,
    ElementOutput,  LayoutC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80, // valid for SM80+ (Orin SM87 compatible)
    cutlass::gemm::GemmShape<128, 64, 64>,      // Threadblock tile
    cutlass::gemm::GemmShape<64, 64, 64>,       // Warp tile
    cutlass::gemm::GemmShape<16, 8, 8>,         // MMA op tile (Tensor Cores)
    EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    2 // stages
>;

void run_cutlass_fp16fp32RowRowRow(const void* A, const void* B, void* C,
                                   int64_t M, int64_t N, int64_t K,
                                   int64_t lda, int64_t ldb, int64_t ldc,
                                   float alpha, float beta, hipStream_t stream) {
  typename Gemm::Arguments args(
      {int(M), int(N), int(K)},
      {reinterpret_cast<ElementInputA const*>(A), int(lda)},
      {reinterpret_cast<ElementInputB const*>(B), int(ldb)},
      {reinterpret_cast<ElementOutput const*>(C), int(ldc)}, // C source for beta
      {reinterpret_cast<ElementOutput*>(C), int(ldc)},
      {alpha, beta});

  Gemm op;
  cutlass::Status st = op.initialize(args, stream);
  if (st != cutlass::Status::kSuccess) {
    throw std::runtime_error("CUTLASS initialize failed");
  }
  st = op(stream);
  if (st != cutlass::Status::kSuccess) {
    throw std::runtime_error("CUTLASS run failed");
  }
}

} // namespace lucidia_gemm
